#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

// Cuda function
__global__ void add(int n, float *one, float *two)
{
  int index = threadIdx.x;
  int jump_length = blockDim.x;
  for (int i = index; i < n; i += jump_length)
      one[i] = one[i] + two[i];
}

int main()
{
  int count = 1000000;
  float *one, *two;

  // Allocate memory using cuda calls
  hipMallocManaged(&one, count*sizeof(float));
  hipMallocManaged(&two, count*sizeof(float));

 // fill arrays with values
  for (int i = 0; i < count; i++)
  {
    one[i] = 1.0f;
    two[i] = 1.0f;
  }

  // Call cuda function for computation
  add<<<1, 256>>>(count, x, y);

  // Wait for Cuda operations to complete
  hipDeviceSynchronize();

  // Release memory using cuda calls
  hipFree(x);
  hipFree(y);
  
  return 0;
}
